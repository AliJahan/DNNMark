#include "hip/hip_runtime.h"
// The MIT License (MIT)
// 
// Copyright (c) 2016 Northeastern University
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in 
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "kernels.h"
#include <stdio.h>
#include <iostream>

namespace dnnmark {

__global__ void BCMForwardKernel(Complex *fft_w,
                                 Complex *fft_x,
                                 Complex *y,
                                 int p, int q, int k) {
  // Dimension of W after FFT is p * q * k (k is floor(n/2)+1)
  // Dimension of X after FFT is n * q * k (k is floor(n/2)+1)
  // Dimension of Y is n * p * k (k is floor(n/2)+1)
  int k_idx = threadIdx.x;
  int p_tid = threadIdx.y;
  int p_bid = blockIdx.y;
  int n_idx = blockIdx.z;
  int p_idx = p_bid * blockDim.y + p_tid;

  extern __shared__ Complex shared_mem[];

  int y_idx = n_idx * p * k + p_idx * k + k_idx;

  if (p_tid == 0) {
    for (int q_idx = 0; q_idx < q; q_idx++) {
      int x_idx = n_idx * q * k + q_idx * k + k_idx;
      shared_mem[q_idx * k + k_idx].x = fft_x[x_idx].x;
      shared_mem[q_idx * k + k_idx].y = fft_x[x_idx].y;
    }
  }
  __syncthreads();


  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int q_idx = 0; q_idx < q; q_idx++) {
    int share_mem_idx = q_idx * k + k_idx;
    int w_idx = p_idx * q * k + q_idx * k + k_idx;
    temp.x += fft_w[w_idx].x * shared_mem[share_mem_idx].x -
                 fft_w[w_idx].y * shared_mem[share_mem_idx].y;
    temp.y += fft_w[w_idx].x * shared_mem[share_mem_idx].y +
               fft_w[w_idx].y * shared_mem[share_mem_idx].x;
  }
  y[y_idx] = temp;

}

void BCMForward(Complex *fft_w, Complex *fft_x, Complex *y,
                int n, int p, int q, int k) {

  int block_size = (k - 1) * 2;
  int tid_p = 1024 / block_size > p ? p : 1024 / block_size; // must be power of 2
  int bid_p = p / tid_p;
  dim3 block_dim(k, tid_p, 1);
  dim3 grid_dim(1, bid_p, n);
  
  size_t shared_mem_size = q * k * sizeof(Complex);
  BCMForwardKernel<<<grid_dim, block_dim, shared_mem_size>>>(fft_w, fft_x, y, p, q, k);
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
      std::cout << "Error: " << hipGetErrorString(err) << std::endl;
}


__global__ void BCMBackwardWeightKernel(Complex *fft_dy,
                                        Complex *fft_x, Complex *dw,
                                        int n, int p, int q, int k) {
  // Dimension of dY after FFT is p * n * k (k is floor(n/2)+1)
  // Dimension of X after FFT is q * n * k (k is floor(n/2)+1)
  // Dimension of dW after this kernel is p * q * k (k is floor(n/2)+1)
  int k_idx = threadIdx.x;
  int q_tid = threadIdx.y;
  int q_bid = blockIdx.y;
  int p_idx = blockIdx.z;
  int q_idx = q_bid * blockDim.y + q_tid;

  extern __shared__ Complex shared_mem[];

  int dw_idx = p_idx * q * k + q_idx * k + k_idx;

  if (q_tid == 0) {
    for (int n_idx = 0; n_idx < n; n_idx++) {
      int dy_idx = n_idx * p * k + p_idx * k + k_idx;
      shared_mem[n_idx * k + k_idx].x = fft_dy[dy_idx].x;
      shared_mem[n_idx * k + k_idx].y = fft_dy[dy_idx].y;
    }
  }
  __syncthreads();

  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int n_idx = 0; n_idx < n; n_idx++) {
    int share_mem_idx = n_idx * k + k_idx;
    int x_idx = n_idx * q * k + q_idx * k + k_idx;
    temp.x += fft_x[x_idx].x * shared_mem[share_mem_idx].x -
                 fft_x[x_idx].y * shared_mem[share_mem_idx].y;
    temp.y += fft_x[x_idx].x * shared_mem[share_mem_idx].y -
               fft_x[x_idx].y * shared_mem[share_mem_idx].x;
  }
  dw[dw_idx] = temp;
}

void BCMBackwardWeight(Complex *fft_dy, Complex *fft_x, Complex *dw,
                int n, int p, int q, int k) {
  int block_size = (k - 1) * 2;
  int tid_q = 1024 / block_size > q ? q : 1024 / block_size; // must be power of 2
  int bid_q = q / tid_q;
  dim3 block_dim(k, tid_q, 1);
  dim3 grid_dim(1, bid_q, p);

  // Shared memory is the limitation
  size_t shared_mem_size = n * k * sizeof(Complex);
  BCMBackwardWeightKernel<<<grid_dim, block_dim, shared_mem_size>>>(fft_dy, fft_x, dw, n, p, q, k);
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
      std::cout << "Error: " << hipGetErrorString(err) << std::endl;
}

__global__ void BCMBackwardDataKernel(Complex *fft_dy,
                                        Complex *fft_w, Complex *dx,
                                        int n, int p, int q, int k) {
  // Dimension of dY after FFT is p * n * k (k is floor(n/2)+1)
  // Dimension of W after FFT is p * q * k (k is floor(n/2)+1)
  // Dimension of dX after this kernel is n * q * k (k is floor(n/2)+1)
  int k_idx = threadIdx.x;
  int q_tid = threadIdx.y;
  int q_bid = blockIdx.y;
  int n_idx = blockIdx.z;
  int q_idx = q_bid * blockDim.y + q_tid;

  extern __shared__ Complex shared_mem[];

  int dx_idx = n_idx * q * k + q_idx * k + k_idx;

  if (q_tid == 0) {
    for (int p_idx = 0; p_idx < p; p_idx++) {
      int dy_idx = n_idx * p * k + p_idx * k + k_idx;
      shared_mem[p_idx * k + k_idx].x = fft_dy[dy_idx].x;
      shared_mem[p_idx * k + k_idx].y = fft_dy[dy_idx].y;
    }
  }
  __syncthreads();

  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int p_idx = 0; p_idx < p; p_idx++) {
    int share_mem_idx = p_idx * k + k_idx;
    int w_idx = p_idx * q * k + q_idx * k + k_idx;
    temp.x += fft_w[w_idx].x * shared_mem[share_mem_idx].x -
                 fft_w[w_idx].y * shared_mem[share_mem_idx].y;
    temp.y += fft_w[w_idx].x * shared_mem[share_mem_idx].y -
               fft_w[w_idx].y * shared_mem[share_mem_idx].x;
  }
  dx[dx_idx] = temp;
}

void BCMBackwardData(Complex *fft_dy, Complex *fft_w, Complex *dx,
                int n, int p, int q, int k) {
  int block_size = (k - 1) * 2;
  int tid_q = 1024 / block_size > q ? q : 1024 / block_size; // must be power of 2
  int bid_q = q / tid_q;
  dim3 block_dim(k, tid_q, 1);
  dim3 grid_dim(1, bid_q, n);

  size_t shared_mem_size = p * k * sizeof(Complex);
  BCMBackwardDataKernel<<<grid_dim, block_dim, shared_mem_size>>>(fft_dy, fft_w, dx, n, p, q, k);
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
      std::cout << "Error: " << hipGetErrorString(err) << std::endl;
}

}
