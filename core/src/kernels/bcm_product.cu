#include "hip/hip_runtime.h"
// The MIT License (MIT)
// 
// Copyright (c) 2016 Northeastern University
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in 
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "kernels.h"
#include <stdio.h>
#include <iostream>

namespace dnnmark {

__global__ void BCMProductForwardKernel(Complex *fft_w, Complex *fft_x, Complex *y) {
  // Dimension of W after FFT is p * q * k (k is floor(n/2)+1)
  // Dimension of X after FFT is n * q * k (k is floor(n/2)+1)
  // Dimension of Y is n * p * q * k (k is floor(n/2)+1)
  int n = gridDim.z;
  int p = gridDim.y;
  int q = gridDim.x;
  int k = blockDim.x;
  int k_idx = threadIdx.x;
  int q_idx = blockIdx.x;
  int p_idx = blockIdx.y;
  int n_idx = blockIdx.z;
  int w_idx = p_idx * q * k + q_idx * k + k_idx;
  int x_idx = n_idx * q * k + q_idx * k + k_idx;
  int y_idx = n_idx * p * q * k + p_idx * q * k + q_idx * k + k_idx;

  y[y_idx].x = fft_w[w_idx].x * fft_x[x_idx].x -
               fft_w[w_idx].y * fft_x[x_idx].y;
  y[y_idx].y = fft_w[w_idx].x * fft_x[x_idx].y +
               fft_w[w_idx].y * fft_x[x_idx].x;

}

void BCMProductForward(Complex *fft_w, Complex *fft_x, Complex *y,
                int n, int p, int q, int k) {
  dim3 block_dim(k, 1, 1);
  dim3 grid_dim(q, p, n);
  BCMProductForwardKernel<<<grid_dim, block_dim>>>(fft_w, fft_x, y);
}

__global__ void BCMProductBackwardWeightKernel(Complex *fft_dy,
                                               Complex *fft_x, Complex *dw) {
  // Dimension of dY after FFT is n * p * k (k is floor(n/2)+1)
  // Dimension of X after FFT is n * q * k (k is floor(n/2)+1)
  // Dimension of dW after this kernel is n * p * q * k (k is floor(n/2)+1)
  int n = gridDim.z;
  int p = gridDim.y;
  int q = gridDim.x;
  int k = blockDim.x;
  int k_idx = threadIdx.x;
  int q_idx = blockIdx.x;
  int p_idx = blockIdx.y;
  int n_idx = blockIdx.z;
  int dy_idx = n_idx * p * k + p_idx * k + k_idx;
  int x_idx = n_idx * q * k + q_idx * k + k_idx;
  int dw_idx = n_idx * p * q * k + p_idx * q * k + q_idx * k + k_idx;

  dw[dw_idx].x = fft_dy[dy_idx].x * fft_x[x_idx].x -
               fft_dy[dy_idx].y * fft_x[x_idx].y;
  dw[dw_idx].y = fft_dy[dy_idx].x * (0 - fft_x[x_idx].y) -
               fft_dy[dy_idx].y * fft_x[x_idx].x;

}

void BCMProductBackwardWeight(Complex *fft_dy, Complex *fft_x, Complex *dw,
                int n, int p, int q, int k) {
  dim3 block_dim(k, 1, 1);
  dim3 grid_dim(q, p, n);
  BCMProductBackwardWeightKernel<<<grid_dim, block_dim>>>(fft_dy, fft_x, dw);
}

__global__ void BCMProductBackwardDataKernel(Complex *fft_dy,
                                             Complex *fft_w, Complex *dx) {
  // Dimension of dY after FFT is n * p * k (k is floor(n/2)+1)
  // Dimension of W after FFT is p * q * k (k is floor(n/2)+1)
  // Dimension of dX after this kernel is n * p * q * k (k is floor(n/2)+1)
  int n = gridDim.z;
  int p = gridDim.y;
  int q = gridDim.x;
  int k = blockDim.x;
  int k_idx = threadIdx.x;
  int q_idx = blockIdx.x;
  int p_idx = blockIdx.y;
  int n_idx = blockIdx.z;
  int dy_idx = n_idx * p * k + p_idx * k + k_idx;
  int w_idx = p_idx * q * k + q_idx * k + k_idx;
  int dx_idx = n_idx * p * q * k + p_idx * q * k + q_idx * k + k_idx;

  dx[dx_idx].x = fft_dy[dy_idx].x * fft_w[w_idx].x -
               fft_dy[dy_idx].y * fft_w[w_idx].y;
  dx[dx_idx].y = fft_dy[dy_idx].x * (0 - fft_w[w_idx].y) -
               fft_dy[dy_idx].y * fft_w[w_idx].x;

}

void BCMProductBackwardData(Complex *fft_dy, Complex *fft_w, Complex *dx,
                int n, int p, int q, int k) {
  dim3 block_dim(k, 1, 1);
  dim3 grid_dim(q, p, n);
  BCMProductBackwardDataKernel<<<grid_dim, block_dim>>>(fft_dy, fft_w, dx);
}

__global__ void BCMProductForwardOptimizedKernel(Complex *fft_w,
                                                 Complex *fft_x,
                                                 Complex *y,
                                                 int p, int q, int k) {
  // Dimension of W after FFT is p * q * k (k is floor(n/2)+1)
  // Dimension of X after FFT is n * q * k (k is floor(n/2)+1)
  // Dimension of Y is n * p * q * k (k is floor(n/2)+1)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  int n_idx = blockIdx.z;
  int p_idx = blockIdx.y;
  int q_idx = idx / k;
  int k_idx = idx % k;

  if (idx < (k * q)) {
    int y_idx = n_idx * p * q * k + p_idx * q * k + idx;
    int w_idx = p_idx * q * k + q_idx * k + k_idx;
    int x_idx = n_idx * q * k + q_idx * k + k_idx;

    y[y_idx].x = fft_w[w_idx].x * fft_x[x_idx].x -
                 fft_w[w_idx].y * fft_x[x_idx].y;
    y[y_idx].y = fft_w[w_idx].x * fft_x[x_idx].y +
                 fft_w[w_idx].y * fft_x[x_idx].x;
  }

}


void BCMProductForwardOptimized(Complex *fft_w, Complex *fft_x, Complex *y,
                int n, int p, int q, int k, int tb_size) {
  int block_size = (k * q + tb_size -1 ) / tb_size;
  dim3 block_dim(tb_size, 1, 1);
  dim3 grid_dim(block_size, p, n);
  BCMProductForwardOptimizedKernel<<<grid_dim, block_dim>>>(fft_w, fft_x, y, p, q, k);
}


__global__ void BCMProductBackwardWeightOptimizedKernel(Complex *fft_dy,
                                               Complex *fft_x, Complex *dw,
                                               int q, int n, int k) {
  // Dimension of dY after FFT is p * n * k (k is floor(n/2)+1)
  // Dimension of X after FFT is q * n * k (k is floor(n/2)+1)
  // Dimension of dW after this kernel is p * q * n * k (k is floor(n/2)+1)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  int p_idx = blockIdx.z;
  int q_idx = blockIdx.y;
  int n_idx = idx / k;
  int k_idx = idx % k;

  if (idx < (n * k)) {
    int dw_idx = p_idx * q * n * k + p_idx * n * k + idx;
    int dy_idx = p_idx * n * k + n_idx * k + k_idx;
    int x_idx = q_idx * n * k + n_idx * k + k_idx;

    dw[dw_idx].x = fft_dy[dy_idx].x * fft_x[x_idx].x -
                 fft_dy[dy_idx].y * fft_x[x_idx].y;
    dw[dw_idx].y = fft_dy[dy_idx].x * (0 - fft_x[x_idx].y) -
                 fft_dy[dy_idx].y * fft_x[x_idx].x;
  }
}

void BCMProductBackwardWeightOptimized(Complex *fft_dy, Complex *fft_x, Complex *dw,
                int n, int p, int q, int k, int tb_size) {
  int block_size = (n * k + tb_size -1 ) / tb_size;
  dim3 block_dim(tb_size, 1, 1);
  dim3 grid_dim(block_size, q, p);
  BCMProductBackwardWeightOptimizedKernel<<<grid_dim, block_dim>>>(fft_dy, fft_x, dw, q, n, k);
}

__global__ void BCMProductBackwardDataOptimizedKernel(Complex *fft_dy,
                                               Complex *fft_w, Complex *dx,
                                               int q, int p, int k) {
  // Dimension of dY after FFT is n * p * k (k is floor(n/2)+1)
  // Dimension of X after FFT is q * p * k (k is floor(n/2)+1)
  // Dimension of dW after this kernel is n * q * p * k (k is floor(n/2)+1)
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int idx = bid * blockDim.x + tid;
  int n_idx = blockIdx.z;
  int q_idx = blockIdx.y;
  int p_idx = idx / k;
  int k_idx = idx % k;

  if (idx < (p * k)) {
    int dx_idx = n_idx * q * p * k + q_idx * p * k + idx;
    int dy_idx = n_idx * p * k + p_idx * k + k_idx;
    int w_idx = q_idx * p * k + p_idx * k + k_idx;

    dx[dx_idx].x = fft_dy[dy_idx].x * fft_w[w_idx].x -
                 fft_dy[dy_idx].y * fft_w[w_idx].y;
    dx[dx_idx].y = fft_dy[dy_idx].x * (0 - fft_w[w_idx].y) -
                 fft_dy[dy_idx].y * fft_w[w_idx].x;
  }
}

void BCMProductBackwardDataOptimized(Complex *fft_dy, Complex *fft_w, Complex *dx,
                int n, int p, int q, int k, int tb_size) {
  int block_size = (p * k + tb_size -1 ) / tb_size;
  dim3 block_dim(tb_size, 1, 1);
  dim3 grid_dim(block_size, q, n);
  BCMProductBackwardDataOptimizedKernel<<<grid_dim, block_dim>>>(fft_dy, fft_w, dx, q, p, k);
}

}
