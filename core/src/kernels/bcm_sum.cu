#include "hip/hip_runtime.h"
// The MIT License (MIT)
// 
// Copyright (c) 2016 Northeastern University
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
// 
// The above copyright notice and this permission notice shall be included in 
// all copies or substantial portions of the Software.
// 
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "kernels.h"

namespace dnnmark {

__global__ void BCMSumForwardKernel(Real *x, Real *y, int q) {
  // Dimension of X is n * p * q * k
  // Dimension of Y is n * q * k
  // Sum over q
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  y[y_idx] = 0;
  Real temp;
  temp = 0;
  for (int i = 0; i < q; i++) {
    int x_idx = blockIdx.x * q * k + i * k + threadIdx.x;
    temp += x[x_idx];
  }
  y[y_idx] = temp;

}

void BCMSumForward(Real *x, Real *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(n * p, 1, 1);
  BCMSumForwardKernel<<<grid_dim, block_dim>>>(x, y, q);
}

__global__ void BCMSumForwardKernel(Complex *x, Complex *y, int q) {
  // Dimension of X is n * p * q * k (k is floor(n/2)+1)
  // Dimension of Y is n * q * k (k is floor(n/2)+1)
  // Sum over q
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int i = 0; i < q; i++) {
    int x_idx = blockIdx.x * q * k + i * k + threadIdx.x;
    temp.x += x[x_idx].x;
    temp.y += x[x_idx].y;
  }
  y[y_idx].x = temp.x;
  y[y_idx].y = temp.y;

}

void BCMSumForward(Complex *x, Complex *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(n * p, 1, 1);
  BCMSumForwardKernel<<<grid_dim, block_dim>>>(x, y, q);
}

__global__ void BCMSumBackwardWeightKernel(Real *x, Real *y, int n) {
  // Dimension of X is n * p * q * k
  // Dimension of Y is p * q * k
  // Sum over n
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  Real temp = 0;
  for (int i = 0; i < n; i++) {
    int x_idx = i * gridDim.x * k + blockIdx.x * k + threadIdx.x;
    temp += x[x_idx];
  }
  y[y_idx] = temp;

}

void BCMSumBackwardWeight(Real *x, Real *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(p * q, 1, 1);
  BCMSumBackwardWeightKernel<<<grid_dim, block_dim>>>(x, y, n);
}

__global__ void BCMSumBackwardWeightKernel(Complex *x, Complex *y, int n) {
  // Dimension of X is n * p * q * k (k is floor(n/2)+1)
  // Dimension of Y is p * q * k (k is floor(n/2)+1)
  // Sum over n
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int i = 0; i < n; i++) {
    int x_idx = i * gridDim.x * k + blockIdx.x * k + threadIdx.x;
    temp.x += x[x_idx].x;
    temp.y += x[x_idx].y;
  }
  y[y_idx].x = temp.x;
  y[y_idx].y = temp.y;
}

void BCMSumBackwardWeight(Complex *x, Complex *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(p * q, 1, 1);
  BCMSumBackwardWeightKernel<<<grid_dim, block_dim>>>(x, y, n);
}

__global__ void BCMSumBackwardDataKernel(Real *x, Real *y, int p, int q) {
  // Dimension of X is n * p * q * k
  // Dimension of Y is n * q * k
  // Sum over p
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  Real temp = 0;
  int n_idx = blockIdx.x / q;
  int q_idx = blockIdx.x % q;
  for (int i = 0; i < p; i++) {
    int x_idx = n_idx * p * q * k + i * q * k + q_idx * k + threadIdx.x;
    temp += x[x_idx];
  }
  y[y_idx] = temp;

}

void BCMSumBackwardData(Real *x, Real *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(n * q, 1, 1);
  BCMSumBackwardDataKernel<<<grid_dim, block_dim>>>(x, y, p, q);
}

__global__ void BCMSumBackwardDataKernel(Complex *x, Complex *y, int p, int q) {
  // Dimension of X is n * p * q * k (k is floor(n/2)+1)
  // Dimension of Y is n * q * k (k is floor(n/2)+1)
  // Sum over p
  int k = blockDim.x;
  int y_idx = blockIdx.x * k + threadIdx.x;
  Complex temp;
  temp.x = 0;
  temp.y = 0;
  int n_idx = blockIdx.x / q;
  int q_idx = blockIdx.x % q;
  for (int i = 0; i < p; i++) {
    int x_idx = n_idx * p * q * k + i * q * k + q_idx * k + threadIdx.x;
    temp.x += x[x_idx].x;
    temp.y += x[x_idx].y;
  }
  y[y_idx].x = temp.x;
  y[y_idx].y = temp.y;
}

void BCMSumBackwardData(Complex *x, Complex *y, int n, int p, int q, int k) {
  dim3 block_dim(k, 1 , 1);
  dim3 grid_dim(n * q, 1, 1);
  BCMSumBackwardDataKernel<<<grid_dim, block_dim>>>(x, y, p, q);
}

__global__ void BCMSumBackwardWeightO2Kernel(Complex *x, Complex *y,
                                int q, int n, int k) {
  // Dimension of X is p * q * n * k (k is floor(n/2)+1)
  // Dimension of Y is p * q * k (k is floor(n/2)+1)
  // Sum over n
  int y_idx = blockIdx.y * q * k + blockIdx.x * q + threadIdx.x;
  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int i = 0; i < n; i++) {
    int x_idx = blockIdx.y * q * n * k + blockIdx.x * n * k + i * k + threadIdx.x;
    temp.x += x[x_idx].x;
    temp.y += x[x_idx].y;
  }
  y[y_idx].x = temp.x;
  y[y_idx].y = temp.y;
}

void BCMSumBackwardWeightO2(Complex *x, Complex *y,
                            int n, int p, int q, int k) {
  dim3 block_dim(k, 1, 1);
  dim3 grid_dim(q, p, 1);
  BCMSumBackwardWeightO2Kernel<<<grid_dim, block_dim>>>(x, y, q, n, k);
}

__global__ void BCMSumBackwardDataO2Kernel(Complex *x, Complex *y,
                                int q, int p, int k) {
  // Dimension of X is n * q * p * k (k is floor(n/2)+1)
  // Dimension of Y is n * q * k (k is floor(n/2)+1)
  // Sum over n
  int y_idx = blockIdx.y * q * k + blockIdx.x * k + threadIdx.x;
  Complex temp;
  temp.x = 0;
  temp.y = 0;
  for (int i = 0; i < p; i++) {
    int x_idx = blockIdx.y * q * p * k + blockIdx.x * p * k + i * k + threadIdx.x;
    temp.x += x[x_idx].x;
    temp.y += x[x_idx].y;
  }
  y[y_idx].x = temp.x;
  y[y_idx].y = temp.y;
}

void BCMSumBackwardDataO2(Complex *x, Complex *y,
                            int n, int p, int q, int k) {
  dim3 block_dim(k, 1, 1);
  dim3 grid_dim(q, n, 1);
  BCMSumBackwardDataO2Kernel<<<grid_dim, block_dim>>>(x, y, q, p, k);
}

}
